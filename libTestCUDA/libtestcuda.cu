#include "hip/hip_runtime.h"
#include "libtestcuda.h"
#include <stdio.h>
#include <hip/hip_vector_types.h>

__global__ void cu_hoge(int tick, uchar4 *bitmap)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	// 連続的になるように...
	float theta = tick / 60.0f * 2.0f * M_PI;
	float theta_x = x / 60.0f * 2.0f * M_PI;
	float theta_y = y / 60.0f * 2.0f * M_PI;
	float r = fabs(sin(theta + theta_x));
	float g = fabs(cos(theta + theta_y));
	float b = fabs(sin(theta + theta_x) * cos(theta + theta_y));

	bitmap[offset].x = (unsigned char)(r * 255);
	bitmap[offset].y = (unsigned char)(g * 255);
	bitmap[offset].z = (unsigned char)(b * 255);
	bitmap[offset].w = 255;
}

static void hoge(int w, int h, int tick, cudamem_t bitmap)
{
	dim3 threads(8, 8); // 64スレッド/1グリッド
	dim3 grids(w / 8, h / 8); // 各ピクセルに1スレッドが割り振られる
	cu_hoge<<<grids, threads>>>(tick, (uchar4 *)bitmap);
}

cudamem_t cuda_malloc(int len)
{
	cudamem_t mem = nullptr;
	hipMalloc((void **)&mem, len);
	return mem;
}

void cuda_free(cudamem_t mem)
{
	hipFree(mem);
}

void cuda_memcpy_htoh(void *dst_h, void *src_h, int len)
{
	hipMemcpy(dst_h, src_h, len, hipMemcpyHostToHost);
}

void cuda_memcpy_dtoh(void *dst_h, void *src_d, int len)
{
	hipMemcpy(dst_h, src_d, len, hipMemcpyDeviceToHost);
}

void cuda_memcpy_htod(void *dst_d, void *src_h, int len)
{
	hipMemcpy(dst_d, src_h, len, hipMemcpyHostToDevice);
}

void cuda_memcpy_dtod(void *dst_d, void *src_d, int len)
{
	hipMemcpy(dst_d, src_d, len, hipMemcpyDeviceToDevice);
}

__global__ void cu_init_cudaimage(uint8_t *p)
{
	int i = blockIdx.x;
	int j = threadIdx.x;
	p[i * 9 + j] = (i + 1) * (j + 1);
}

CUDAIMAGE_API api;

extern "C" CUDAIMAGE_API const *init_cudaimage(int n)
{
	if (n != sizeof(CUDAIMAGE_API)) return nullptr;

	uint8_t table[81];
	uint8_t *mem;
	hipMalloc((void **)&mem, 81);
	dim3 b(9);
	dim3 t(9);
	cu_init_cudaimage<<<b,t>>>(mem);
	hipMemcpy(table, mem, 81, hipMemcpyDeviceToHost);
	hipFree(mem);

	// test
	for (int i = 0; i < 9; i++) {
		for (int j = 0; j < 9; j++) {
			if (table[9 * i + j] != (i + 1) * (j + 1)) {
				return nullptr;
			}
		}
	}

	api.malloc = cuda_malloc;
	api.free = cuda_free;
	api.memcpy_htoh = cuda_memcpy_htoh;
	api.memcpy_dtoh = cuda_memcpy_dtoh;
	api.memcpy_htod = cuda_memcpy_htod;
	api.memcpy_dtod = cuda_memcpy_dtod;
	api.hoge = hoge;
	return &api;
}
